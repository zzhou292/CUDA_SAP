#include <stdio.h>
#include <hip/hip_runtime.h>


#define CudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

float* CudaAllocate(int64_t size)
{
    float* result;
    hipMalloc(&result, sizeof(float) * size);
    CudaCheckErrors("Failed to allocate device occupancy grid");
    return result;
}
